#include "hip/hip_runtime.h"
#include "juliaset.cuh"
#include <iostream>
#include <cmath>

__device__ __host__ juliaset::juliaset(){}

__device__ __host__ void juliaset::display(){
//std::cout << zRe << " +i"<<zIm<<std::endl;
}

__device__ __host__ real juliaset::transform(unsigned width,unsigned pixelNumber)
{
    real Z = (pixelNumber - (width * 0.5)) / (0.25* width);
    return Z;
}

__device__ __host__ real juliaset::getMagnitude(const juliaset& Z){
    return std::sqrt((Z.zRe*Z.zRe) + (Z.zIm * Z.zIm));
}

