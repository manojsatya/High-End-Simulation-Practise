#include <iostream>
#include <vector>
#include <sys/time.h>
#include <string>
#include "type.h"
#include "juliaset.cuh"
#include "lodepng.h"
#include "juliaAlgorithm.cuh"

using namespace std;

void checkError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

double getSeconds() {
    timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec * (double)tp.tv_usec * 1e-6);
}

int main(int argc, const char *argv[]){

    if(argc > 4 || argc < 1){std::cout << "Please enter only threadx and thready" << std::endl; abort();}
    int threadsX = std::atoi(argv[1]);
    int threadsY = std::atoi(argv[2]);
    const char* filename1 = "julia1.png"; //const char* filename2 = "julia2.png" ;
    const long long ARRAY_SIZE = 2048 * 2048 * 3;
    const long long ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);
    unsigned char* h_image_out = new unsigned char[ARRAY_SIZE];
    double startTime,stopTime;

    //unsigned char * d_image_out;
    unsigned char* d_image_out;// = new unsigned char[ARRAY_SIZE];

    checkError(hipMalloc((void**)&d_image_out, ARRAY_BYTES));

    startTime = getSeconds();
    dim3 threadsPerBlock(threadsX,threadsY);
    dim3 numBlocks(2048/threadsPerBlock.x,2048/threadsPerBlock.y);
    // Kernel launch
    juliaAlgorithm <<<numBlocks , threadsPerBlock>>> (d_image_out);
    hipError_t(hipPeekAtLastError());
    hipDeviceSynchronize();
    stopTime = getSeconds();
    std::cout << "time :" << (stopTime-startTime) * 1e-3 << "ms" << std::endl;
    checkError(hipMemcpy(h_image_out, d_image_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    lodepng::encode(filename1, h_image_out, 2048, 2048,LCT_RGB,8);
    checkError(hipFree(d_image_out));

    return 0;
}
