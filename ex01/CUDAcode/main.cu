#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "type.h"
#include "juliaset.cuh"
#include "lodepng.h"
//#include "Timer.h"
//#include "juliaAlgorithm.cuh"

using namespace std;

void checkError(hipError_t err) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}


void encodeToPng(const std::string& filename,unsigned char * image, unsigned width, unsigned height,
                 LodePNGColorType colortype = LCT_PALETTE,unsigned bitdepth = 4){

      //Encode the image
      unsigned error = lodepng::encode(filename, image, width, height);

      //if there's an error, display it
      if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}

__global__ void juliaAlgorithm(unsigned width, unsigned height, juliaset& C,unsigned char* d_image_in,
                               unsigned char* d_image_out){

    //int ix = blockIdx.x * blockDim.x + threadIdx.x;
    //int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int ix = threadIdx.x;
    int iy = threadIdx.y;
    //int x = blockIdx.x; int y = blockIdx.y;
    //int offset = x + y * gridDim.x;
    //std::cout << "Image is being created. Please wait ..... " << std::endl;
    //td::vector<unsigned char> image;
    //for(unsigned i = 0;i< width;++i)
      //  for(unsigned j = 0;j< height;++j){
            juliaset Z,d;
            real zreal = d.transform( width,ix );
            real zimag = d.transform( height,iy);

            juliaset zold(zreal,zimag);
            unsigned iter;


            for(iter = 1; iter < 400; iter++){
            Z = (zold * zold) + C;
            zold = Z;
            if(Z.getMagnitude(Z) > 2) {break;}
            }
                //buffer to image
                //d_image_out[4 * width * iy + 4 * ix + 0] = iter;
                //d_image_out[4 * width * iy + 4 * ix + 1] = iter;
                //d_image_out[4 * width * iy + 4 * ix + 2] = iter;
                //d_image_out[4 * width * iy + 4 * ix + 3] = iter;
            d_image_out[3*ix + 3*2048 * iy +0] = 255 * iter;
            //std::cout << "Iteration Number:" << iter << std::endl;
        //}
    //return image;
}

int main(){

    real cRe,cIm;
    unsigned width = 2048,height = 2048;
    const char* filename1 = "julia1.png"; //const char* filename2 = "julia2.png" ;
    const int ARRAY_SIZE = width * height;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    //Vector Image for buffering
    //std::vector<unsigned char> h_image_in,h_image_out;

    //std::cout << "I am here" << std::endl;
    //unsigned char h_image_in[ARRAY_SIZE],h_image_out[ARRAY_SIZE];
    unsigned char* h_image_in = new unsigned char[ARRAY_SIZE];
    unsigned char* h_image_out = new unsigned char[ARRAY_SIZE];
    //for(int i = 0;i<ARRAY_SIZE;i++){
        //h_image_in[] = 0;
        //h_image_out[] = 0;
    //}
    //unsigned char* h_image_in;
    //unsigned char* h_image_out;
    //image.resize(width * height * 4);
    //std::cout << "I am here" << std::endl;
    //hespa::Timer time; // Start timer

    // Change Real and Imaginary according to the problem
    cRe = -0.8; cIm = 0.2;
    juliaset C(cRe,cIm);

    //Pointers for objects to host(h) and device(d)
    //juliaset *h_C_in; h_C_in = &C;
    //h_image_in[ARRAY_SIZE]; // host image
    //h_image_out[ARRAY_SIZE]; // device image

    //juliaset *d_C_in;
    //juliaset *d_C_out;

    //std::vector<unsigned char> d_image_in;
    //std::vector<unsigned char> d_image_out;
        // declare GPU pointers
    //unsigned char * d_image_in;
    unsigned char* d_image_in = new unsigned char[ARRAY_SIZE];
    //std::cout << "I am here" << std::endl;
    //unsigned char * d_image_out;
    unsigned char* d_image_out = new unsigned char[ARRAY_SIZE];

    checkError(hipMalloc((void**)&d_image_in, ARRAY_BYTES));
    //std::cout << "I am here" << std::endl;
    checkError(hipMalloc((void**)&d_image_out, ARRAY_BYTES));
    //std::cout << "I am here" << std::endl;

    // Allocating GPU memory
    //hipMalloc((void**) &d_image_in, ARRAY_BYTES);
    //hipMalloc((void**) &d_image_out, ARRAY_BYTES);

    //Transferrring array to GPU
    checkError(hipMemcpy(d_image_in, h_image_in, ARRAY_BYTES, hipMemcpyHostToDevice));

    std::cout << "I am here" << std::endl;
    dim3 threadsPerBlock(16,16);

    juliaAlgorithm <<<1,threadsPerBlock>>> (width,height,C, d_image_in, d_image_out); // Kernel launch
    hipError_t(hipPeekAtLastError());
    //juliaset *h_C_out; h_C_out = &C;

    checkError(hipMemcpy(h_image_out, d_image_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
    //juliaAlgorithm<<< 1 , 1 >>>(width,height,C,image);


    //std::cout << "First image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename1,h_image_out,width,height); // write image

    checkError(hipFree(d_image_in));
    checkError(hipFree(d_image_out));








    /*time.reset();

    cRe = 0.0; cIm = 0.8;
    juliaset D(cRe,cIm);

    image = juliaAlgorithm(width,height,D,image);
    std::cout << "Second image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename2,image,width,height); // write image*/

    return 0;
}
