#include <iostream>
#include <vector>
#include "type.h"
#include "juliaset.h"
#include "lodepng.h"
#include "Timer.h"
#include "juliaAlgorithm.cuh"

using namespace std;

void encodeToPng(const std::string& filename,std::vector<unsigned char>& image, unsigned width, unsigned height,
                 LodePNGColorType colortype = LCT_PALETTE,unsigned bitdepth = 4){

      //Encode the image
      unsigned error = lodepng::encode(filename, image, width, height);

      //if there's an error, display it
      if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}

int main(){

    real cRe,cIm;
    unsigned width = 2048,height = 2048;
    const char* filename1 = "julia1.png";const char* filename2 = "julia2.png" ;
    const int ARRAY_SIZE = width * height;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(real);

    //Vector Image for buffering
    std::vector<unsigned char> image;
    image.resize(width * height * 4);

    hespa::Timer time; // Start timer

    // Change Real and Imaginary according to the problem
    cRe = -0.8; cIm = 0.2;
    juliaset C(cRe,cIm);

    //Pointers for objects to host(h) and device(d)
    juliaset *h_C_in;
    juliaset *h_C_out;
    juliaset *d_C_in;
    juliaset *d_C_out;

    // Allocating GPU memory
    hipMalloc((void**) &d_C_in, ARRAY_BYTES);
    hipMalloc((void**) &d_C_out, ARRAY_BYTES);

    //Transferrring array to GPU
    hipMemcpy(d_C_in, h_C_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    image = juliaAlgorithm<<<1,ARRAY_SIZE>>>(width,height,C,image); // Kernel launch

    hipMemcpy(h_C_out, d_C_out, ARRAY_BYTES, hipMemcpyDeviceToHost);



    std::cout << "First image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename1,image,width,height); // write image

    hipFree(d_C_in);
    hipFree(d_C_out);








    /*time.reset();

    cRe = 0.0; cIm = 0.8;
    juliaset D(cRe,cIm);

    image = juliaAlgorithm(width,height,D,image);
    std::cout << "Second image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename2,image,width,height); // write image*/

    return 0;
}
