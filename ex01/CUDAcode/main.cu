#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "type.h"
#include "juliaset.cuh"
#include "lodepng.h"
#include "Timer.h"
//#include "juliaAlgorithm.cuh"

using namespace std;


void encodeToPng(const std::string& filename,unsigned char * image, unsigned width, unsigned height,
                 LodePNGColorType colortype = LCT_PALETTE,unsigned bitdepth = 4){

      //Encode the image
      unsigned error = lodepng::encode(filename, image, width, height);

      //if there's an error, display it
      if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}

__global__ void juliaAlgorithm(unsigned width, unsigned height, juliaset& C,unsigned char*  d_image_in,
                               unsigned char* d_image_out){

    //std::cout << "Image is being created. Please wait ..... " << std::endl;
    //td::vector<unsigned char> image;
    for(unsigned i = 0;i< width;++i)
        for(unsigned j = 0;j< height;++j){
            juliaset Z,d;
            real zreal = d.transform( width,i );
            real zimag = d.transform( height,j);

            juliaset zold(zreal,zimag);
            unsigned iter;


            for(iter = 1; iter < 400; iter++){
            Z = (zold * zold) + C;
            zold = Z;
            if(Z.getMagnitude(Z) > 2) {break;}
            }
                //buffer to image
                d_image_out[4 * width * j + 4 * i + 0] = iter;
                d_image_out[4 * width * j + 4 * i + 1] = iter;
                d_image_out[4 * width * j + 4 * i + 2] = iter;
                d_image_out[4 * width * j + 4 * i + 3] = iter;
            //std::cout << "Iteration Number:" << iter << std::endl;
        }
    //return image;
}

int main(){

    real cRe,cIm;
    unsigned width = 2048,height = 2048;
    const char* filename1 = "julia1.png"; //const char* filename2 = "julia2.png" ;
    const int ARRAY_SIZE = width * height;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(real);

    //Vector Image for buffering
    //std::vector<unsigned char> h_image_in,h_image_out;
    unsigned char h_image_in[ARRAY_SIZE],h_image_out[ARRAY_SIZE];
    //image.resize(width * height * 4);

    hespa::Timer time; // Start timer

    // Change Real and Imaginary according to the problem
    cRe = -0.8; cIm = 0.2;
    juliaset C(cRe,cIm);

    //Pointers for objects to host(h) and device(d)
    //juliaset *h_C_in; h_C_in = &C;
    //h_image_in[ARRAY_SIZE]; // host image
    //h_image_out[ARRAY_SIZE]; // device image

    //juliaset *d_C_in;
    //juliaset *d_C_out;

    //std::vector<unsigned char> d_image_in;
    //std::vector<unsigned char> d_image_out;
        // declare GPU pointers
    unsigned char * d_image_in;
    unsigned char * d_image_out;

    // Allocating GPU memory
    hipMalloc((void**) &d_image_in, ARRAY_BYTES);
    hipMalloc((void**) &d_image_out, ARRAY_BYTES);

    //Transferrring array to GPU
    hipMemcpy(d_image_in, h_image_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    juliaAlgorithm <<<1,ARRAY_SIZE>>> (width,height,C, d_image_in, d_image_out); // Kernel launch
    //juliaset *h_C_out; h_C_out = &C;

    hipMemcpy(h_image_out, d_image_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    //juliaAlgorithm<<< 1 , 1 >>>(width,height,C,image);


    std::cout << "First image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename1,h_image_out,width,height); // write image

    hipFree(d_image_in);
    hipFree(d_image_out);








    /*time.reset();

    cRe = 0.0; cIm = 0.8;
    juliaset D(cRe,cIm);

    image = juliaAlgorithm(width,height,D,image);
    std::cout << "Second image Executed in :" << time.elapsed() << "seconds" << std::endl;
    encodeToPng(filename2,image,width,height); // write image*/

    return 0;
}
